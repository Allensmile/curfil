#if 0
#######################################################################################
# The MIT License

# Copyright (c) 2014       Hannes Schulz, University of Bonn  <schulz@ais.uni-bonn.de>
# Copyright (c) 2013       Benedikt Waldvogel, University of Bonn <mail@bwaldvogel.de>
# Copyright (c) 2008-2009  Sebastian Nowozin                       <nowozin@gmail.com>

# Permission is hereby granted, free of charge, to any person obtaining a copy
# of this software and associated documentation files (the "Software"), to deal
# in the Software without restriction, including without limitation the rights
# to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
# copies of the Software, and to permit persons to whom the Software is
# furnished to do so, subject to the following conditions:
# 
# The above copyright notice and this permission notice shall be included in all
# copies or substantial portions of the Software.
# 
# THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
# IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
# FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
# AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
# LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
# OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
# SOFTWARE.
#######################################################################################
#endif

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

namespace curfil {
namespace gpu {

__global__
void setup_kernel(int seed, hiprandState *state) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, id, 0, &state[id]);
}

__global__
void generate_uniform_kernel(hiprandState* state, unsigned int* result) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int count = 0;
    float x;

    /* Copy state to local memory for efficiency */
    hiprandState localState = state[id];

    /* Generate pseudo-random uniforms */
    for (int n = 0; n < 10000; n++) {
        x = hiprand_uniform(&localState);
        /* Check if > .5 */
        if (x > .5) {
            count++;
        }
    }

    /* Copy state back to global memory */
    state[id] = localState;

    /* Store results */
    result[id] += count;
}

}
}
